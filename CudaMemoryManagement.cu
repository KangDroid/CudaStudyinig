
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void changeValue(int *val) {
    *val = 50;
}

__global__ void changeValueArr(int *val) {
    val[blockIdx.x] = 40;
}

template <typename DEFTYPE>
class CudaMemoryManagement {
private:
    DEFTYPE* arr_store[100];
    int arr_store_ctr;
public:
    CudaMemoryManagement() {
        this->arr_store_ctr = 0;
    }
    ~CudaMemoryManagement() {
        // Free memory here?
        for (int i = 0; i < arr_store_ctr; i++) {
            hipFree(arr_store[i]);
        }
    }

    // Create memory area on GPU - side
    DEFTYPE* createMemory(DEFTYPE originalValue) {
        DEFTYPE* device_value;
        hipMalloc((void **)&device_value, sizeof(DEFTYPE));
        hipMemcpy(device_value, &originalValue, sizeof(DEFTYPE), hipMemcpyHostToDevice);
        // Register to arr_store
        this->arr_store[arr_store_ctr++] = device_value;
        return device_value;
    }

    // Overload function for previous createMemory (For 1d array implementation)
    DEFTYPE* createMemory(DEFTYPE* originalValue, int length) {
        DEFTYPE* device_value;
        hipMalloc((void **)&device_value, sizeof(DEFTYPE) * length);
        hipMemcpy(device_value, &originalValue, sizeof(DEFTYPE) * length, hipMemcpyHostToDevice);
        // Register to arr_store
        this->arr_store[arr_store_ctr++] = device_value;
        return device_value;
    }

    // Get the result from device_ptr(GPU's memory)
    void getResult(DEFTYPE *device_ptr, DEFTYPE *host_ptr) {
        hipMemcpy(host_ptr, device_ptr, sizeof(DEFTYPE), hipMemcpyDeviceToHost);
    }

    // Overload function for previous getResult (For 1d array implementation)
    void getResult(DEFTYPE *device_ptr, DEFTYPE *host_ptr, int length) {
        hipMemcpy(host_ptr, device_ptr, sizeof(DEFTYPE) * length, hipMemcpyDeviceToHost);
    }
};

int main(void) {
    CudaMemoryManagement<int> cmm;
    int arr_host[10];
    for (int i = 0; i < 10; i++) {
        arr_host[i] = rand()%10;
    }
    int *dev_arr = cmm.createMemory(arr_host, 10);
    changeValueArr<<<10, 1>>>(dev_arr);
    cmm.getResult(dev_arr, arr_host, 10);
    for (int i = 0; i < 10; i++) {
        cout << arr_host[i] << endl;
    }
    return 0;
}